#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <string>
#include <stdexcept>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>

namespace py = pybind11;

__global__ void PDV_main(
    const float* __restrict__ rates_a,
    const float* __restrict__ rates_b,
    float* __restrict__ results,
    size_t numT,
    size_t num_neurons,
    size_t N_WARPS
);

__global__ void cosD_main(
    float* __restrict__ rates_a,
    float* __restrict__ rates_b,
    float* __restrict__ results,
    size_t numT,
    size_t num_neurons,
    size_t N_WARPS
);

py::array_t<float> compute_D_gpu(
    py::array_t<float, py::array::c_style | py::array::forcecast> rates_a,
    py::array_t<float, py::array::c_style | py::array::forcecast> rates_b,
    int selection
)
{
    size_t num_neurons = rates_a.shape()[1]; // assuming both stimulus were presented to the same ntwrk configuration
    size_t num_runs = rates_a.shape()[0];

    size_t numT = 256;
    const size_t N_WARPS = numT/32;
    dim3 numBlocks(num_runs);
    dim3 threadsPerBlock(numT);

    const float* h_a = rates_a.data();
    const float* h_b = rates_b.data();
    size_t input_size = num_runs * num_neurons * sizeof(float);

    size_t output_size = num_runs * sizeof(float);

    float* g_a = nullptr;
    float* g_b = nullptr;
    float* g_res = nullptr;

    hipMalloc(&g_a, input_size), hipMalloc(&g_b, input_size), hipMalloc(&g_res, output_size);
    hipMemcpy(g_a, h_a, input_size, hipMemcpyHostToDevice);
    hipMemcpy(g_b, h_b, input_size, hipMemcpyHostToDevice);

    switch(selection)
    {
        case 0: {
            size_t sharedMem_size = N_WARPS * sizeof(float);
            PDV_main<<<numBlocks, threadsPerBlock, sharedMem_size>>>(g_a, g_b, g_res, numT, num_neurons, N_WARPS);
            break;
        }
        case 1: {
            size_t sharedMem_size = 3 * N_WARPS * sizeof(float);
            cosD_main<<<numBlocks, threadsPerBlock, sharedMem_size>>>(g_a, g_b, g_res, numT, num_neurons, N_WARPS);
            break;
        }
        default: throw std::invalid_argument("Invalid selection of distance measure for the population-wise rate distance computation");
    }

    auto h_res = py::array_t<float>(num_runs);
    float* h_res_ptr = h_res.mutable_data();

    hipMemcpy(h_res_ptr, g_res, output_size, hipMemcpyDeviceToHost);

    hipFree(g_a), hipFree(g_b), hipFree(g_res);

    return h_res;
};

__global__ void PDV_main( // ! not safe for any block dim other than 256 ! when this will work well, will add dynamic branching with templates
    const float* __restrict__ rates_a,
    const float* __restrict__ rates_b,
    float* __restrict__ results,
    size_t numT,
    size_t num_neurons,
    size_t N_WARPS
)
{
    extern __shared__ float b_sum[];
    const size_t tid = threadIdx.x;

    size_t i = tid + blockIdx.x*num_neurons;
    float p_sum = 0.0f;
    // unrolled loop for summing distances, controlling which threads get which neurons
    p_sum = (rates_a[i] - rates_b[i]) * (rates_a[i] - rates_b[i]);
    i += numT;
    p_sum += (rates_a[i] - rates_b[i]) * (rates_a[i] - rates_b[i]);
    i += numT;
    p_sum += (rates_a[i] - rates_b[i]) * (rates_a[i] - rates_b[i]);
    i += numT;
    size_t next_neuron_idx = tid + 3*numT;
    if(next_neuron_idx < num_neurons)
    {
        p_sum += (rates_a[i] - rates_b[i]) * (rates_a[i] - rates_b[i]);
    }

    constexpr unsigned int MASK = 0xffffffff;
    #pragma unroll
    for(size_t offset = 16; offset > 0; offset /= 2)
    {
        p_sum += __shfl_down_sync(MASK, p_sum, offset);
    }

    if(tid % 32 == 0)
    {
        b_sum[tid / 32] = p_sum;
    }
    __syncthreads();

    float final_sum = 0.0f;
    
    if(tid == 0)
    {
        #pragma unroll
        for(size_t i{0}; i < N_WARPS; i++)
        {
            final_sum += b_sum[i];
        }

        results[blockIdx.x] = sqrtf(final_sum);
    }
};

__global__ void cosD_main(
    float* __restrict__ rates_a,
    float* __restrict__ rates_b,
    float* __restrict__ results,
    size_t numT,
    size_t num_neurons,
    size_t N_WARPS
)
{
    extern __shared__ float b_mem[];
    float* b_dotprod = b_mem;
    float* b_mA = b_dotprod + N_WARPS;
    float* b_mB = b_mA + N_WARPS;

    const size_t tid = threadIdx.x;

    float p_dotprod = 0.0f;
    float p_mA = 0.0f;
    float p_mB = 0.0f;
    // unrolling isnt really useful, as the computing time is almost null compared to the cudamemcpy time anyway
    // and keeping the loop allows for doing the comp on various pops
    for (size_t i = tid; i < num_neurons; i += numT) {
        size_t g_idx = blockIdx.x * num_neurons + i;
        
        p_dotprod += rates_a[g_idx] * rates_b[g_idx];
        p_mA += rates_a[g_idx] * rates_a[g_idx];
        p_mB += rates_b[g_idx] * rates_b[g_idx];
    }

    constexpr unsigned int MASK = 0xffffffff;
    #pragma unroll
    for(size_t offset = 16; offset > 0; offset /= 2)
    {
        p_dotprod += __shfl_down_sync(MASK, p_dotprod, offset);
        p_mA += __shfl_down_sync(MASK, p_mA, offset);
        p_mB += __shfl_down_sync(MASK, p_mB, offset);
    }
    
    if(tid % 32 == 0)
    {
        b_dotprod[tid / 32] = p_dotprod;
        b_mA[tid / 32] = p_mA;
        b_mB[tid / 32] = p_mB;
    }
    __syncthreads();

    if(tid == 0)
    {
        float final_dotprod = 0.0f;
        float final_mA = 0.0f;
        float final_mB = 0.0f;

        #pragma unroll
        for(size_t i{0}; i < N_WARPS; i++)
        {
            final_dotprod += b_dotprod[i];
            final_mA += b_mA[i];
            final_mB += b_mB[i];
        }

        final_mA = sqrtf(final_mA);
        final_mB = sqrtf(final_mB);
        final_dotprod /= (final_mA * final_mB);

        final_dotprod = fminf(1.0f, fmaxf(-1.0f, final_dotprod));

        results[blockIdx.x] = (2.0f * acosf(final_dotprod)) / M_PIf;
    }
};

PYBIND11_MODULE(pdv_cuda, m)
{
    m.doc() = "PDV computation between 2 arrays of rates (runs x neurons) on gpu";
    m.def("compute_D_gpu", &compute_D_gpu, "computes PDV on gpu for 2 rates arrays");
}
